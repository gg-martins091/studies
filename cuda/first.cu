
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <assert.h>

#define N 10000000
#define MAX_ERR 1e-6

__global__ void vector_add(float *out, float *a, float *b, int n) {
    // 1 thread per item add
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid < n){
        out[tid] = a[tid] + b[tid];
    }
}

__global__ void old_vector_add(float *out, float *a, float *b, int n) {
    int i = threadIdx.x;
    int stride = blockDim.x;
    while(i < n) {
        out[i] = a[i] + b[i];
        i += stride;
    }
}

int main(){
    float *a, *b, *out; 
    float *ad, *bd, *outd;

    // Allocate memory
    a   = (float*)malloc(sizeof(float) * N);
    b   = (float*)malloc(sizeof(float) * N);
    out = (float*)malloc(sizeof(float) * N);

    // Initialize array
    for(int i = 0; i < N; i++){
        a[i] = 1.0f;
        b[i] = 2.0f;
    }


    hipMalloc((void**)&ad, sizeof(float) * N);
    hipMalloc((void**)&bd, sizeof(float) * N);
    hipMalloc((void**)&outd, sizeof(float) * N);

    hipMemcpy(ad, a, sizeof(float) * N, hipMemcpyHostToDevice);
    hipMemcpy(bd, b, sizeof(float) * N, hipMemcpyHostToDevice);
    /* for (int i = 0; i < N; i++) { */
    /*     printf("%f ", a[i]); */
    /* } */

    int block_size = 256; // 256 threads per block
    int grid_size = ((N+block_size)/block_size); // to have 1 thread per item
    // 1 thread per operation very fast 740us
    vector_add<<<grid_size,block_size>>>(outd, ad, bd, N);
    hipMemcpy(out, outd, sizeof(float) * N, hipMemcpyDeviceToHost);

    // 256 threads with 256 stride: N/256 operations per thread 14ms
    vector_add<<<grid_size,block_size>>>(outd, ad, bd, N);

    old_vector_add<<<1,256>>>(outd, ad, bd, N);
    hipMemcpy(out, outd, sizeof(float) * N, hipMemcpyDeviceToHost);

    // Verification
    for(int i = 0; i < N; i++){
        assert(fabs(out[i] - a[i] - b[i]) < MAX_ERR);
    }

    printf("out[0] = %f\n", out[0]);
    printf("PASSED\n");

    free(a);
    hipFree(ad);
    free(b);
    hipFree(bd);
    free(out);
    hipFree(outd);
}
